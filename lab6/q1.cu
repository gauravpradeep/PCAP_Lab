#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void conv1d(float *N, float *M, float *P, int mwidth, int width){
	int tid=blockIdx.x*blockDim.x + threadIdx.x;

	float res=0;
	int start=tid-(mwidth/2);
	for(int i=0;i<mwidth;i++)
	{
		if(start+i>=0 && start+i<width)
			res+=N[start+i]*M[i];
	}
	P[tid]=res;
}

int main(){
 	int N,M;
 	
 	printf("Enter size of array");
 	scanf("%d",&N);
 	float *h_N = (float*)malloc(N*sizeof(float));
 	printf("Enter size of kernel");
 	scanf("%d",&M);
 	float *h_M = (float*)malloc(N*sizeof(float));
 	float *h_P = (float*)malloc(N*sizeof(float));

 	float *d_N, *d_M, *d_P;

 	hipMalloc((void**)&d_N,N*sizeof(float));
 	hipMalloc((void**)&d_M,M*sizeof(float));
 	hipMalloc((void**)&d_P,N*sizeof(float));
 	printf("Enter elements of N");
 	for(int i=0;i<N;i++)
 		scanf("%f",h_N+i);
 	printf("Enter elements of mask");
 	for(int i=0;i<M;i++)
 		scanf("%f",h_M+i);

 	hipMemcpy(d_N,h_N,N*sizeof(int),hipMemcpyHostToDevice);
 	hipMemcpy(d_M,h_M,M*sizeof(int),hipMemcpyHostToDevice);
 	conv1d<<<1,N>>>(d_N, d_M, d_P, M, N);
 	hipMemcpy(h_P,d_P,N*sizeof(int),hipMemcpyDeviceToHost);
 	printf("Convolution result \n");
 	for(int i=0;i<N;i++)
 		printf("%f ",*(h_P+i));
 	printf("\n");

 	hipFree(d_N);
 	hipFree(d_M);
 	hipFree(d_P);

}