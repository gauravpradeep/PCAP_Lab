#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void oddEven(int *A, int n)
{
	int tid=blockIdx.x*blockDim.x +threadIdx.x;
	if(tid%2==1 && tid+1<n)
	{
		if(A[tid]>A[tid+1])
		{
			int temp=A[tid];
			A[tid]=A[tid+1];
			A[tid+1]=temp;
		}
	}
}

__global__ void evenOdd(int *A, int n)
{
	int tid=blockIdx.x*blockDim.x +threadIdx.x;
	if(tid%2==0 && tid+1<n)
	{
		if(A[tid]>A[tid+1])
		{
			int temp=A[tid];
			A[tid]=A[tid+1];
			A[tid+1]=temp;
		}
	}
}

int main(){
 	int N;
 	
 	printf("Enter size of array");
 	scanf("%d",&N);
 	int *h_A = (int*)malloc(N*sizeof(int));

 	int *d_A;

 	hipMalloc((void**)&d_A,N*sizeof(int));
 	printf("Enter elements of A");
 	for(int i=0;i<N;i++)
 		scanf("%d",h_A+i);

 	hipMemcpy(d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice);
 	for(int i=0;i<N/2;i++)
 	{
 		oddEven<<<1,N>>>(d_A,N);
 		evenOdd<<<1,N>>>(d_A,N);
 	}
 	hipMemcpy(h_A,d_A,N*sizeof(int),hipMemcpyDeviceToHost);
 	printf("Sorted array \n");
 	for(int i=0;i<N;i++)
 		printf("%d ",*(h_A+i));
 	printf("\n");

 	hipFree(d_A);
}