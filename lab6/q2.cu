#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void selsort(int *A, int *O, int n)
{
	int tid=blockIdx.x*blockDim.x + threadIdx.x;
	int pos=0;
	int ele=A[tid];
	for(int i=0;i<n;i++)
	{
		if(A[i]<ele || (A[i]==ele && i<tid))
			pos++;
	}
	O[pos]=ele;
}

int main(){
 	int N;
 	
 	printf("Enter size of array");
 	scanf("%d",&N);
 	int *h_A = (int*)malloc(N*sizeof(int));
 	int *h_O = (int*)malloc(N*sizeof(int));

 	int *d_A, *d_O;

 	hipMalloc((void**)&d_A,N*sizeof(int));
 	hipMalloc((void**)&d_O,N*sizeof(int));

 	printf("Enter elements of A");
 	for(int i=0;i<N;i++)
 		scanf("%d",h_A+i);

 	hipMemcpy(d_A,h_A,N*sizeof(int),hipMemcpyHostToDevice);
 	selsort<<<1,N>>>(d_A,d_O,N);
 	hipMemcpy(h_O,d_O,N*sizeof(int),hipMemcpyDeviceToHost);
 	printf("Sorted array \n");
 	for(int i=0;i<N;i++)
 		printf("%d ",*(h_O+i));
 	printf("\n");

 	hipFree(d_A);
}