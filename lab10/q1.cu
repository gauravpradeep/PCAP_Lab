#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CUDA kernel for matrix multiplication
__global__ void matrixMulKernel(float *d_A, float *d_B, float *d_C, int width) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < width && Col < width) {
        float Cvalue = 0.0;
        for (int k = 0; k < width; ++k) {
            Cvalue += d_A[Row * width + k] * d_B[k * width + Col];
        }
        d_C[Row * width + Col] = Cvalue;
    }
}

void matrixMultiplication(float *h_A, float *h_B, float *h_C, int width) {
    int size = width * width * sizeof(float);
    
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16); // 16x16 threads per block
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y); // Grid size

    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, width);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int width = 3; // Example width of the matrices
    int size = width * width;
    float *h_A = new float[size];
    float *h_B = new float[size];
    float *h_C = new float[size];

    // Initialize matrices with some values
    for (int i = 0; i < size; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i);
    }

    matrixMultiplication(h_A, h_B, h_C, width);

    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            std::cout << h_C[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}