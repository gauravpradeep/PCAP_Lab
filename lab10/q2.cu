#include <hip/hip_runtime.h>

#include <stdio.h>

#define MAX_KERNEL_SIZE 1024

__constant__ float d_M[MAX_KERNEL_SIZE];

__global__ void conv1d(float *N, float *P, int mwidth, int width) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    float res = 0;
    int start = tid - (mwidth / 2);
    for (int i = 0; i < mwidth; i++) {
        if (start + i >= 0 && start + i < width)
            res += N[start + i] * d_M[i];
    }
    P[tid] = res;
}

int main() {
    int N, M;

    printf("Enter size of array: ");
    scanf("%d", &N);
    float *h_N = (float*)malloc(N * sizeof(float));
    printf("Enter size of kernel: ");
    scanf("%d", &M);
    if (M > MAX_KERNEL_SIZE) {
        printf("Kernel size exceeds maximum allowed size of %d\n", MAX_KERNEL_SIZE);
        return 1;
    }
    float *h_M = (float*)malloc(M * sizeof(float));
    float *h_P = (float*)malloc(N * sizeof(float));

    float *d_N, *d_P;

    hipMalloc((void**)&d_N, N * sizeof(float));
    hipMalloc((void**)&d_P, N * sizeof(float));
    printf("Enter elements of N: ");
    for (int i = 0; i < N; i++)
        scanf("%f", h_N + i);
    printf("Enter elements of mask: ");
    for (int i = 0; i < M; i++)
        scanf("%f", h_M + i);

    hipMemcpy(d_N, h_N, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_M), h_M, M * sizeof(float));
    conv1d<<<(N + 255) / 256, 256>>>(d_N, d_P, M, N);
    hipMemcpy(h_P, d_P, N * sizeof(float), hipMemcpyDeviceToHost);
    printf("Convolution result: \n");
    for (int i = 0; i < N; i++)
        printf("%f ", *(h_P + i));
    printf("\n");

    hipFree(d_N);
    hipFree(d_P);
    free(h_N);
    free(h_M);
    free(h_P);

    return 0;
}