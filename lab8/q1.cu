#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void add1(int* a, int* b, int* c, int m, int n) {
    int rid = threadIdx.y;
    int cid = threadIdx.x;
    c[n*rid + cid] = a[n*rid + cid]+b[n*rid + cid];
}

__global__ void add2(int* a, int* b, int* c, int m, int n){
    int rid=threadIdx.x;
    for(int i=0;i<n;i++)
        c[rid*n + i] = a[rid*n + i] + b[rid*n + i];
}
__global__ void add3(int* a, int* b, int* c, int m, int n){
    int cid=threadIdx.x;
    for(int i=0;i<n;i++)
        c[i*n+cid] = a[i*n+cid] + b[i*n+cid];
}


int main() {
    int M,N;
    
    printf("Enter no of rows of matrix");
    scanf("%d",&M);
    printf("Enter no of columns of matrix");
    scanf("%d",&N);

    int h_A[M][N],h_B[M][N],h_C[M][N];
    int *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A,M*N*sizeof(int));
    hipMalloc((void**)&d_B,M*N*sizeof(int));
    hipMalloc((void**)&d_C,M*N*sizeof(int));

    printf("Enter elements of A");
    
    for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
            scanf("%d",&h_A[i][j]);
    }

    printf("Enter elements of B");
    for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
            scanf("%d",&h_B[i][j]);
    }

    hipMemcpy(d_A,h_A,M*N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_B,h_B,M*N*sizeof(int),hipMemcpyHostToDevice);
    // dim3 blockDim(N, M);
    // add1<<<1,blockDim>>>(d_A,d_B,d_C,M,N);
// --------------------------------

    // dim3 blockDim(M,1);
    // add2<<<1,blockDim>>>(d_A,d_B,d_C,M,N);
// --------------------------------

    dim3 blockDim(N,1);  
    add3<<<1,blockDim>>>(d_A,d_B,d_C,M,N);
// --------------------------------


    hipMemcpy(h_C,d_C,M*N*sizeof(int),hipMemcpyDeviceToHost);
    for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
            printf("%d ",h_C[i][j]);
        printf("\n");
    }

    return 0;
}