#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void multiplyKernel_rowwise(int *a, int *b, int *c, int wa, int wb) {
    int ridA = threadIdx.x;  // Each thread computes one row
    if (ridA < wa) {
        for (int cidB = 0; cidB < wb; cidB++) {
            int sum = 0;
            for (int k = 0; k < wa; k++) {
                sum += a[ridA * wa + k] * b[k * wb + cidB];
            }
            c[ridA * wb + cidB] = sum;
        }
    }
}

__global__ void multiplyKernel_colwise(int *a, int *b, int *c, int ha, int wa, int wb) {
    int cidB = threadIdx.x;  // Each thread computes one column
    if (cidB < wb) {
        for (int ridA = 0; ridA < ha; ridA++) {
            int sum = 0;
            for (int k = 0; k < wa; k++) {
                sum += a[ridA * wa + k] * b[k * wb + cidB];
            }
            c[ridA * wb + cidB] = sum;
        }
    }
}

__global__ void multiplyKernel_elementwise(int *a, int *b, int *c, int wa, int wb) {
    int ridA = threadIdx.y;
    int cidB = threadIdx.x;
    
    if (ridA < wa && cidB < wb) {
        int sum = 0;
        for (int k = 0; k < wa; k++) {
            sum += a[ridA * wa + k] * b[k * wb + cidB];
        }
        c[ridA * wb + cidB] = sum;
    }
}

int main() {
    int n, m, a, b;

    printf("Enter n and m for A(nxm): ");
    scanf("%d %d", &n, &m);

    int A[n][m];
    printf("Enter A:\n");
    for (int i = 0; i < n; i++)
        for (int j = 0; j < m; j++)
            scanf("%d", &A[i][j]);

    printf("Enter a and b for B(axb): ");
    scanf("%d %d", &a, &b);

    if (m != a) {
        printf("Matrix multiplication not possible. m must equal a.\n");
        return -1;
    }

    int B[a][b], C[n][b], D[n][b], E[n][b];

    printf("Enter B:\n");
    for (int i = 0; i < a; i++)
        for (int j = 0; j < b; j++)
            scanf("%d", &B[i][j]);

    int *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeof(int) * n * m);
    hipMalloc((void**)&d_B, sizeof(int) * a * b);
    hipMalloc((void**)&d_C, sizeof(int) * n * b);

    hipMemcpy(d_A, A, sizeof(int) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(int) * a * b, hipMemcpyHostToDevice);

    // Set grid and block dimensions dynamically
    dim3 blockDim1(n, 1);
    multiplyKernel_rowwise<<<1, blockDim1>>>(d_A, d_B, d_C, m, b);
    hipDeviceSynchronize();  // Ensure kernel finishes
    hipMemcpy(C, d_C, sizeof(int) * n * b, hipMemcpyDeviceToHost);

    dim3 blockDim2(1, b);
    multiplyKernel_colwise<<<1, blockDim2>>>(d_A, d_B, d_C, n, m, b);
    hipDeviceSynchronize();
    hipMemcpy(D, d_C, sizeof(int) * n * b, hipMemcpyDeviceToHost);

    dim3 blockDim3(b, n);
    multiplyKernel_elementwise<<<1, blockDim3>>>(d_A, d_B, d_C, m, b);
    hipDeviceSynchronize();
    hipMemcpy(E, d_C, sizeof(int) * n * b, hipMemcpyDeviceToHost);

    printf("Row-wise multiplication (each thread handles a row):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < b; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }

    printf("Column-wise multiplication (each thread handles a column):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < b; j++)
            printf("%d ", D[i][j]);
        printf("\n");
    }

    printf("Element-wise multiplication (each thread handles an element):\n");
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < b; j++)
            printf("%d ", E[i][j]);
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}