#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
__global__ void CUDACountWord(char *text, int textLength, char *word, int wordLength, unsigned int *d_count, int *wordLengths) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int start=0;
    bool match=true;
    for(int i=0;i<tid;i++)
    {
        start+=wordLengths[i];
        start+=1;
    }
    printf("%d %d",tid, wordLengths[tid]);
    if(wordLengths[tid]!=wordLength)
        match=false;
    else{
        for(int i=0;i<wordLength;i++)
        {
            if(text[start+i]!=word[i])
                match=false;
        }
    }
        if (match) {
            atomicAdd(d_count, 1);
        }
}

int main() {
    char text[100];
    char word[10];
    char *d_text, *d_word;
    int *d_wordLengths;
    unsigned int count = 0, result;
    unsigned int *d_count;
    
    printf("Enter a string: ");
    fgets(text,100,stdin);
    int textLength = strlen(text);
    if (text[textLength - 1] == '\n') text[textLength - 1] = '\0'; // Remove newline
    textLength = strlen(text);
    int wordLengths[textLength];
    int len=0,wc=0;
    for(int i=0;i<textLength;i++)
    {
        if(text[i]!=' ' && text[i]!='\0')
            len++;
        else
        {
            wordLengths[wc]=len;
            wc+=1;
            len=0;
        }
    }
    if (len > 0) {
        wordLengths[wc] = len;
        wc += 1;
    }
    printf("Enter word to search: ");
    scanf("%s",word);
    int wordLength = strlen(word);
    wordLength = strlen(word); 

    hipMalloc((void**)&d_text, textLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    hipMalloc((void**)&d_wordLengths, wc* sizeof(int));
    hipMemcpy(d_text, text, textLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_wordLengths, wordLengths, wc * sizeof(int), hipMemcpyHostToDevice);

    CUDACountWord<<<1, wc>>>(d_text, textLength, d_word, wordLength, d_count, d_wordLengths);
    
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    printf("Total occurrences of '%s' = %u\n", word, result);
    
    hipFree(d_text);
    hipFree(d_word);
    hipFree(d_count);
    
    return 0;
}