#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void sinAngles(float *a, float *b){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	*(b+tid)=sin(*(a+tid));	
 }

 int main(){
 	int N;
 	
 	printf("Enter size of array");
 	scanf("%d",&N);
 	int size=N*sizeof(float);
 	float *a = (float*)malloc(size);
 	float *b = (float*)malloc(size);
 	float *d_a, *d_b;
 	hipMalloc((void**)&d_a,size);
 	hipMalloc((void**)&d_b,size);
 	printf("Enter elements of array");
 	for(int i=0;i<N;i++)
 		scanf("%f",a+i);
 	
 	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 	sinAngles<<<1,N>>>(d_a,d_b);
 	hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);
 	printf("N threads result \n");
 	for(int i=0;i<N;i++)
 		printf("%f ",*(b+i));
  	hipFree(d_a);
 	hipFree(d_b);
 	
 }
