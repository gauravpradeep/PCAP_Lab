#include <hip/hip_runtime.h>

#include <stdio.h>
__global__ void vectorAdd(int *a, int *b, int *c){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	*(c+tid)=*(a+tid)+*(b+tid);	
 }

 int main(){
 	int N;
 	
 	printf("Enter size of array");
 	scanf("%d",&N);
 	int *a = (int*)malloc(N*sizeof(int));
 	int *b = (int*)malloc(N*sizeof(int));
 	int *c = (int*)malloc(N*sizeof(int));

 	int *d_a, *d_b, *d_c;
 	int size=N*sizeof(int);
 	hipMalloc((void**)&d_a,size);
 	hipMalloc((void**)&d_b,size);
 	hipMalloc((void**)&d_c,size);
 	printf("Enter elements of A");
 	for(int i=0;i<N;i++)
 		scanf("%d",a+i);
 	printf("Enter elements of B");
 	for(int i=0;i<N;i++)
 		scanf("%d",b+i);

 	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
 	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
 	vectorAdd<<<1,N>>>(d_a,d_b,d_c);
 	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 	printf("N threads result \n");
 	for(int i=0;i<N;i++)
 		printf("%d ",*(c+i));
 	printf("\n");
 	vectorAdd<<<N,1>>>(d_a,d_b,d_c);
 	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 	printf("N blocks result \n");
 	for(int i=0;i<N;i++)
 		printf("%d ",*(c+i));
 	printf("\n");

 	dim3 dimGrid(ceil(N/256),1,1);
 	dim3 dimBlock(256,1,1);

	vectorAdd<<<dimGrid, dimBlock>>>(d_a,d_b,d_c);
 	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
 	printf("256 threads result \n");
 	for(int i=0;i<N;i++)
 		printf("%d ",*(c+i));	
 	printf("\n");

 	hipFree(d_a);
 	hipFree(d_b);
 	hipFree(d_c);

}